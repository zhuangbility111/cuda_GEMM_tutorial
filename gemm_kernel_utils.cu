#include "gemm.h"
#include "gemm_kernel_utils.cuh"

void check_cuda_last(const char* const file, const int line) {
    hipError_t const err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}