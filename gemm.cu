#include "hip/hip_runtime.h"
#include "gemm.h"


void run_perf_test(float *A, float *B, float *C, int M, int N, int K, int warmup, int repeat, dim3 grid, dim3 block, std::string version, gemm_func func) {
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));

    hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, M * N * sizeof(float), hipMemcpyHostToDevice);

    // warm up
    if (version == "cuBLAS") {
        for (int i = 0; i < warmup; i++)
            func(d_A, d_B, d_C, M, N, K);
    } else {
        for (int i = 0; i < warmup; i++)
            func<<<grid, block>>>(d_A, d_B, d_C, M, N, K);
    }

    // performance test
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    if (version == "cuBLAS") {
        for (int i = 0; i < repeat; i++)
            func(d_A, d_B, d_C, M, N, K);
    } else {
        for (int i = 0; i < repeat; i++)
            func<<<grid, block>>>(d_A, d_B, d_C, M, N, K);
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    float avg_time = ms / repeat;

    printf("Test %s: average time of %d runs %.6f ms, %.6f Tflops, percentage of peak %.6f\n", version.c_str(), repeat, avg_time, 2.0 * M * N * K / avg_time / 1e9, 2.0 * M * N * K / avg_time / 1e9 / 19.5 * 100);

    hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


void gemm_cublas(float *d_A, float *d_B, float *d_C, int M, int N, int K) {
    // create cublas handle
    hipblasHandle_t handle;
    checkCublasError(hipblasCreate(&handle), "Failed to create cublas handle");

    // 调用 hipblasSgemm
    float alpha = 1.0f, beta = 0.0f;
    checkCublasError(
        hipblasSgemm(handle,
                    HIPBLAS_OP_T, HIPBLAS_OP_T, // 矩阵 A 和 B 的操作类型：不转置
                    M, N, K,                 // 矩阵 A 的维度 (MxK)，B 的维度 (KxN)，结果 C 的维度 (MxN)
                    &alpha,                  // alpha 系数
                    d_A, K,                  // 矩阵 A 和其主列间距
                    d_B, N,                  // 矩阵 B 和其主列间距
                    &beta,                   // beta 系数
                    d_C, N),                 // 矩阵 C 和其主列间距
        "Failed to call hipblasSgemm");
    
}

// version 1: naive implementation
__global__ void gemm_naive(float *A, float *B, float *C, int M, int N, int K) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; k++) 
            sum += A[row * K + k] * B[k * N +col];
        C[row * N + col] = sum;
    }
}


// version 2: coalesced memory access
__global__ void gemm_coalesced(float *A, float *B, float *C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0.0;
        for (int k = 0; k < K; k++) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

// version 3: blocking (tiling) on shared memory
// block_size must equals blockDim.x equals blockDim.y
__global__ void gemm_shared_mem_blocking(float *A, float *B, float *C, int M, int N, int K) {
    __shared__ float s_A[BLOCK_SIZE * BLOCK_SIZE];
    __shared__ float s_B[BLOCK_SIZE * BLOCK_SIZE];

    int row_in_shared_mem = threadIdx.y;
    int col_in_shared_mem = threadIdx.x;

    float* A_ptr = A + blockIdx.y * BLOCK_SIZE * K;
    float* B_ptr = B + blockIdx.x * BLOCK_SIZE;
    float* C_ptr = C + blockIdx.y * BLOCK_SIZE * N + blockIdx.x * BLOCK_SIZE;

    float tmp = 0.0;
    for (int s_block_idx = 0; s_block_idx < K; s_block_idx += BLOCK_SIZE) {
        // load data from global memory to shared memory
        s_A[row_in_shared_mem * BLOCK_SIZE + col_in_shared_mem] = A_ptr[row_in_shared_mem * K + col_in_shared_mem];
        s_B[row_in_shared_mem * BLOCK_SIZE + col_in_shared_mem] = B_ptr[row_in_shared_mem * N + col_in_shared_mem];

        __syncthreads();

        A_ptr += BLOCK_SIZE;
        B_ptr += BLOCK_SIZE * N;

        // compute
        for (int k = 0; k < BLOCK_SIZE; k++) {
            tmp += s_A[row_in_shared_mem * BLOCK_SIZE + k] * s_B[k * BLOCK_SIZE + col_in_shared_mem];
        }

        __syncthreads();
    }
    
    C_ptr[row_in_shared_mem * BLOCK_SIZE + col_in_shared_mem] = tmp;
}